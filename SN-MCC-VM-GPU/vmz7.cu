#include "hip/hip_runtime.h"
// Compile: nvcc vmz.cu -o vmz 
// Run:     ./vmz < net.mcc
// daze@acm.org
// max_cooperative_grid_blocks()

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>

// data length definitions

//#define _LONG_MU_

#ifdef _LONG_MU_

#define MUTY long
#define MUMAX LONG_MAX

#else

#define MUTY int
#define MUMAX INT_MAX

#endif

// end of data length definitions

using namespace cooperative_groups; 

#define MATRIX_SIZE(d1,d2,t) ((d1)*(d2)*(sizeof(t)))
#define VECTOR_SIZE(d1,t)    ((d1)*(sizeof(t)))

#define MOFF(i,j,d1,d2) ((d2)*(i)+(j))
#define MELT(x,i,j,d1,d2) (*((x)+MOFF(i,j,d1,d2)))

#define zmax(x,y) (((x)>(y))?(x):(y))
#define zmin(x,y) (((x)<(y))?(x):(y))

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void read_matr_int(int *x,int m,int n)
{
  int i,j;
  for(i=0;i<m;i++)
  {
    for(j=0;j<n;j++)
      scanf("%d",&MELT(x,i,j,m,n));
  }
}

void read_vect_int(int *x,int m)
{
  int i;
  for(i=0;i<m;i++)
  {
    scanf("%d",x+i);
  }
}

void read_vect_long(long *x,int m)
{
  int i;
  for(i=0;i<m;i++)
  {
    scanf("%ld",x+i);
  }
}

void print_matr_int(int *x,int m,int n)
{
  int i,j;
  for(i=0;i<m;i++)
  {
    for(j=0;j<n;j++)
      printf("%10d ",MELT(x,i,j,m,n));
    printf("\n");
  }
}

void print_matr_long(long *x,int m,int n)
{
  int i,j;
  for(i=0;i<m;i++)
  {
    for(j=0;j<n;j++)
      printf("%10ld ",MELT(x,i,j,m,n));
    printf("\n");
  }
}

void print_vect_int(int *x,int m)
{
  int i;
  for(i=0;i<m;i++)
  {
    printf("%d ",x[i]);
  }
  printf("\n");
}

void print_vect_long(long *x,int m)
{
  int i;
  for(i=0;i<m;i++)
  {
    printf("%ld ",x[i]);
  }
  printf("\n");
}


struct type_f {
  MUTY c;
  int t;
  long k;
};

__global__ void run_sn(int m, int n, int mm, int *bi, int *bv, int *di, int *dv, MUTY *mu, MUTY *y, struct type_f *f, long maxk, int dbg)
{
  int pi = threadIdx.x;
  int t = blockIdx.x;
  int step;
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();

  if( pi==0 && t==0 ) (f->k)=0;

  while( (f->k) < maxk )
  {
    // fire_arc
    MELT(y,pi,t,mm,n) = (MELT(bv,pi,t,mm,n)>0)? mu[MELT(bi,pi,t,mm,n)] / MELT(bv,pi,t,mm,n) : 
                        (MELT(bv,pi,t,mm,n)<0)? ((mu[MELT(bi,pi,t,mm,n)]>0)? 0: MUMAX): MUMAX;
    grid.sync();
 
    /* fire_trs
    if(pi==0)
    {
      
      MELT(y,mm,t,mm,n)=MELT(y,0,t,mm,n);
      for(int ppi=1;ppi<mm;ppi++)
        MELT(y,mm,t,mm,n)=zmin(MELT(y,mm,t,mm,n),MELT(y,ppi,t,mm,n));
    }
    grid.sync();*/
    
    // fire_trs, red
    for(step=1; step<mm; step*=2) {
      if(pi%(step*2)==0) {
	if(pi+step<mm) MELT(y,pi,t,mm,n)=zmin(MELT(y,pi,t,mm,n),MELT(y,pi+step,t,mm,n));
      }
      __syncthreads();
    }
//    grid.sync();

    /* choose_f_trs
    if( pi==0 && t==0 )
    {
      int tt;

      (f->c)=0; (f->t)=0;
	 
      for(tt=0; tt<n; tt++)
      {
        if(MELT(y,0,tt,mm,n)>0)
        {
          (f->c)=MELT(y,0,tt,mm,n); // firing multiplicity
          (f->t)=tt;   		     // firing transition number
          break;
        }
      }
    }
    grid.sync();
*/

    // choose_f_trs, red
    
    
      if(pi==0) MELT(y,1,t,mm,n)=t;
      if((t==0) && (pi==0)) (f->c)=0;
      grid.sync();
      for(step=1; ((step<n) && (f->c)==0); step*=2) {
        if(pi==0) {if(t%(step*2)==0) {
          if(MELT(y,0,t,mm,n)>0) {
    	    if(t==0) { (f->c)=MELT(y,0,t,mm,n); (f->t)=MELT(y,1,t,mm,n); }
          } else if(t+step<n) {
            if(MELT(y,0,t+step,mm,n)>0) {
	      MELT(y,0,t,mm,n)=MELT(y,0,t+step,mm,n);
	      MELT(y,1,t,mm,n)=MELT(y,1,t+step,mm,n);
	      if(t==0) { (f->c)=MELT(y,0,t,mm,n); (f->t)=MELT(y,1,t,mm,n); }
            } 
          }
          }
        }
        grid.sync();
      }
//      if(t==0 && pi==0) { (f->c)=MELT(y,0,t,mm,n); (f->t)=MELT(y,1,t,mm,n);}
//      grid.sync();
    // end of choose_f_trs, red
    
    // is somebody firing ?
    if((f->c)>0){ 
      if( t==0 ) // next_mu
      {
        if(MELT(bv,pi,f->t,mm,n)>0) mu[MELT(bi,pi,f->t,mm,n)]-=(f->c)*MELT(bv,pi,f->t,mm,n);
        if(MELT(dv,pi,f->t,mm,n)>0) mu[MELT(di,pi,f->t,mm,n)]+=(f->c)*MELT(dv,pi,f->t,mm,n);
      }
    } else break;
    if( pi==0 && t==0 ) (f->k)++;
    grid.sync();

  } // end of while

} // end of run_sn



int main(int argc, char * argv[])
{
  int m, n, mm;
  int *bi, *bv, *di, *dv;
  MUTY *mu, *y;
  int *d_bi, *d_bv, *d_di, *d_dv;
  MUTY *d_mu, *d_y;
  struct type_f *d_f;
  int dbg=0;
  long maxk=MUMAX;
  struct type_f f;
  double t1, dt;
  
  if(argc>1) dbg=atoi(argv[1]);
  if(argc>2) maxk=atoi(argv[2]);
  
  // read mcc
  
    scanf("%d %d %d\n", &m, &n, &mm);
if(dbg>0)printf("m=%d n=%d mm=%d\n", m, n, mm);
  
  bi=(int *)malloc(MATRIX_SIZE(mm,n,int));
  bv=(int *)malloc(MATRIX_SIZE(mm,n,int));
  dv=(int *)malloc(MATRIX_SIZE(mm,n,int));
  di=(int *)malloc(MATRIX_SIZE(mm,n,int));
  mu=(MUTY *)malloc(VECTOR_SIZE(m,MUTY));
  if(dbg>2) y=(MUTY *)malloc(MATRIX_SIZE(mm,n,MUTY)); else y=(MUTY *)bi; // a trick to have no null
  if( bi==NULL || di==NULL || bv==NULL || dv==NULL || mu==NULL || y==NULL )
  {
    printf("*** error: not enough memory\n");
    exit(3);
  }
  
  read_matr_int(bi,mm,n);
if(dbg>2){
printf("bi:\n");
print_matr_int(bi,mm,n);}
  read_matr_int(bv,mm,n);
if(dbg>2){
printf("bv:\n");
print_matr_int(bv,mm,n);}

  read_matr_int(di,mm,n);
if(dbg>2){
printf("di:\n");
print_matr_int(di,mm,n);}
  read_matr_int(dv,mm,n);
if(dbg>2){
printf("dv:\n");
print_matr_int(dv,mm,n);}

#ifdef _LONG_MU_
  read_vect_long(mu,m);
if(dbg>0){
  printf("initial mu:\n");
  print_vect_long(mu,m);
}
#else
  read_vect_int(mu,m);
if(dbg>0){
  printf("initial mu:\n");
  print_vect_int(mu,m);
}
#endif
 
  // allocate device memory & copy to device
   
  CHECK(hipSetDevice(0));

  int supportsCoopLaunch = 0;
  hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, 0);

  if(!supportsCoopLaunch)
  {
    printf("*** no supportsCoopLaunch\n");
    exit(2);
  }

if(dbg>1) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("multiprocessors: %d\n",deviceProp.multiProcessorCount);
}

  CHECK(hipMalloc((int**)&d_bi, MATRIX_SIZE(mm,n, int)));
  CHECK(hipMalloc((int**)&d_bv, MATRIX_SIZE(mm,n, int)));
  CHECK(hipMalloc((int**)&d_di, MATRIX_SIZE(mm,n, int)));
  CHECK(hipMalloc((int**)&d_dv, MATRIX_SIZE(mm,n, int)));
  CHECK(hipMalloc((MUTY**)&d_mu, VECTOR_SIZE(m, MUTY)));
  CHECK(hipMalloc((MUTY**)&d_y, MATRIX_SIZE(mm,n, MUTY)));
  CHECK(hipMalloc((struct type_f**)&d_f, sizeof(struct type_f) ));
  
  CHECK(hipMemcpy(d_bi, bi, MATRIX_SIZE(mm,n, int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_bv, bv, MATRIX_SIZE(mm,n, int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_di, di, MATRIX_SIZE(mm,n, int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_dv, dv, MATRIX_SIZE(mm,n, int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_mu, mu, VECTOR_SIZE(m, MUTY), hipMemcpyHostToDevice));
  
  // define device grid & block
    
  dim3 block (mm);
  dim3 grid  (n);

  void* kernelArgs[] = { (void*)&m,(void*)& n,(void*)& mm,(void*)& d_bi,(void*)& d_bv,(void*)& d_di,(void*)& d_dv,(void*)& d_mu,(void*)& d_y,(void*)& d_f,(void*)& maxk,(void*)& dbg };

  // run sn on device
//  printf("max_cooperative_grid_blocks(mm)=%d\n",run_sn.max_cooperative_grid_blocks(mm));
  
  t1=seconds();
  //run_sn<<<grid, block>>>(m, n, mm, d_bi, d_bv, d_di, d_dv, d_mu, d_y, d_f, maxk, dbg);
   hipLaunchCooperativeKernel((void*)run_sn, grid, block, kernelArgs);
   CHECK(hipDeviceSynchronize());
  dt=seconds()-t1;
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(&f, d_f, sizeof(struct type_f), hipMemcpyDeviceToHost));

if(dbg>1){
  printf("*** step: %ld, transition %d fired in %ld copies\n", f.k, f.t, (long)f.c);    
}

  // copy from device and print resulting marking
if(dbg>0){      
  CHECK(hipMemcpy(mu, d_mu, VECTOR_SIZE(m,MUTY), hipMemcpyDeviceToHost));  
  printf("final mu:\n");
  #ifdef _LONG_MU_  
    print_vect_long(mu,m);
  #else
    print_vect_int(mu,m);
  #endif
}

if(dbg>2){
  CHECK(hipMemcpy(y, d_y, MATRIX_SIZE(mm,n, MUTY), hipMemcpyDeviceToHost));
  printf("y:\n");
  #ifdef _LONG_MU_                              
    print_matr_long(y,mm,n);
  #else
    print_matr_int(y,mm,n);
  #endif 
}
  
  printf("--- it took %ld steps, time %f s. ---\n",f.k,dt);
  
  // free memory of device and host
  
  CHECK(hipFree(d_bi));
  CHECK(hipFree(d_bv));
  CHECK(hipFree(d_di));
  CHECK(hipFree(d_dv));
  CHECK(hipFree(d_mu));
  CHECK(hipFree(d_y));
  
  free(bi);
  free(bv);
  free(di); 
  free(dv); 
  free(mu); 
  if(dbg>2) free(y);

} // end of main

// daze@acm.org

